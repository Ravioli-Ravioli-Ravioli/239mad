
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <time.h>

#define RNC 100

__global__ void kernel_1t1e(int A[RNC][RNC], int B[RNC][RNC], int C[RNC][RNC], int size){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;
   if (i < size && j < size){
	   A[i][j] = B[i][j] + C[i][j];
   }
}

__global__ void kernel_1t1r(int A[RNC][RNC], int B[RNC][RNC], int C[RNC][RNC], int size){
   int j = blockIdx.y * blockDim.y + threadIdx.y;
   for (int k = 0; k < size; k++){
	if (j < size){
	     A[k][j] = B[k][j] + C[k][j];	
	}
   }
}

__global__ void kernel_1t1c(int A[RNC][RNC], int B[RNC][RNC], int C[RNC][RNC], int size){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   for (int k = 0; k < size; k++){
        if (i < size){
             A[i][k] = B[i][k] + C[i][k]; 
        }
   }
}

int main(void) {
  int nDevices, i, j, B[RNC][RNC], C[RNC][RNC], A[RNC][RNC], (*A_d)[RNC], (*B_d)[RNC], (*C_d)[RNC] ;

//////////////////////////////////////Print device properties

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  MaxThreadPerBlock: %d\n",
           prop.maxThreadsPerBlock);
//    printf("  MultiProcessorCount: %d\n",
//           prop.multiProcessorCount);
//    printf("  ClockRate: %d\n",
//           prop.clockRate);
    printf("  MaxThreadsDim0: %d\n",
           prop.maxThreadsDim[0]);
    printf("  MaxThreadsDim1: %d\n",
           prop.maxThreadsDim[1]);
    printf("  MaxThreadsDim2: %d\n",
           prop.maxThreadsDim[2]);
    printf("  MaxGridSize: %d\n",
           prop.maxGridSize[1]);
//    printf("  Memory Clock Rate (KHz): %d\n",
//           prop.memoryClockRate);
//    printf("  Memory Bus Width (bits): %d\n",
//           prop.memoryBusWidth);
//    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
//           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
/////////////////////////////////////Populate First Matrix
   srand(1);
   for (i = 0; i < RNC; i++){
      for (j = 0; j < RNC; j++) {
         B[i][j] = rand() % 100 + 1; 
         printf("%d  ",B[i][j]);
      }
      printf("\n");
   }
   printf("\n");
/////////////////////////////////////Populate Second Matrix
   for (i = 0; i < RNC; i++){
      for (j = 0; j < RNC; j++) {
         C[i][j] = rand() % 100 + 1;
         printf("%d  ",C[i][j]);
      }
      printf("\n");
   }
   printf("\n");
   printf("===============================");
   printf("\n");
/////////////////////////////////////Allocate memory in the device
   
   hipMalloc((void**) &A_d, (RNC*RNC)*sizeof(int));
   hipMalloc((void**) &B_d, (RNC*RNC)*sizeof(int));
   hipMalloc((void**) &C_d, (RNC*RNC)*sizeof(int));

////////////////////////////////////Mem copy from host to device
   hipMemcpy(A_d, A, (RNC*RNC)*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, (RNC*RNC)*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(C_d, C, (RNC*RNC)*sizeof(int), hipMemcpyHostToDevice);

   dim3 threadsPerBlock(RNC, RNC);
   dim3 numBlocks(RNC / threadsPerBlock.x, RNC / threadsPerBlock.y);

   hipEvent_t start, stop;
   float elapsed = 0;

////////////////////////////////////ThreadAll
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);
   kernel_1t1e<<<numBlocks,threadsPerBlock>>>(A_d, B_d, C_d, RNC);
   hipEventRecord(stop, 0);

   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);
   hipEventDestroy(start);
   hipEventDestroy(stop);
   printf("GPU Run TIme threadsall %.2f ms \n", elapsed);
////////////////////////////////////Thread Row
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);
   kernel_1t1r<<<numBlocks,threadsPerBlock>>>(A_d, B_d, C_d, RNC);
   hipEventRecord(stop, 0);

   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);
   hipEventDestroy(start);
   hipEventDestroy(stop);
   printf("GPU Run TIme threadsrow %.2f ms \n", elapsed);
////////////////////////////////////Thread Column
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);
   kernel_1t1c<<<numBlocks,threadsPerBlock>>>(A_d, B_d, C_d, RNC);
   hipEventRecord(stop, 0);

   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);
   hipEventDestroy(start);
   hipEventDestroy(stop);
   printf("GPU Run TIme threadscol %.2f ms \n", elapsed);
//////////////////////////////////////Mem Copy
   hipMemcpy(A, A_d, (RNC*RNC)*sizeof(int), hipMemcpyDeviceToHost);

/////////////////////////////////////Print matrix A
/*
   for (i = 0; i < RNC; i++){
      for (j = 0; j < RNC; j++) {
         printf("%d  ", A[i][j]);
      }
      printf("\n");
   }
   printf("\n");
*/
/////////////////////////////////////Free up memory
   hipFree(A_d); 
   hipFree(B_d); 
   hipFree(C_d);
}
